#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/gmm_layer.hpp"

#include <opencv2\opencv.hpp>

namespace caffe {

//alpha & sigma exponential
template <typename Dtype>
__global__ void sigmaExp(const int nthreads, const int param_size, int sigma_min, int sigma_max, Dtype* const topdata) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		int vecIdx = index % param_size;

		if (vecIdx == 0)						//alpha
			topdata[index] = exp(topdata[index]);
		else if (vecIdx == (param_size - 1)){	//sigma
			if (topdata[index] > sigma_max) 		topdata[index] = exp((float)sigma_max);
			else if (topdata[index] < sigma_min)	topdata[index] = exp((float)sigma_min);
			else									topdata[index] = exp(topdata[index]);
		}
	}
}

template <typename Dtype>
__global__ void kernel_alpha_max(const int num, const int param_size, const int class_size, const Dtype* data, Dtype* out) {
	CUDA_KERNEL_LOOP(index, num) {
		Dtype maxval = -FLT_MAX;
		for (int i = 0; i < class_size; i++)
			maxval = max(data[(index*class_size*param_size) + i*param_size], maxval);

		out[index] = maxval;
	}
}

template <typename Dtype>
__global__ void kernel_alpha_subtract(const int count,
	const int param_size, const int class_size,
	const Dtype* max, Dtype* data) {
	CUDA_KERNEL_LOOP(index, count) {
		int n = index / class_size;
		data[index*param_size] -= max[n];
	}
}

template <typename Dtype>
__global__ void kernel_alpha_sum(const int num, const int param_size, const int class_size, const Dtype* data, Dtype* out) {
	CUDA_KERNEL_LOOP(index, num) {
		Dtype sum = 0;
		for (int i = 0; i < class_size; i++)
			sum += data[index*class_size*param_size + i*param_size];
		out[index] = sum;
	}
}

template <typename Dtype>
__global__ void kernel_alpha_div(const int count,
	const int param_size, const int class_size,
	const Dtype* sum, Dtype* data) {
	CUDA_KERNEL_LOOP(index, count) {
		int n = index / class_size;
		data[index*param_size] /= sum[n];
	}
}

template <typename Dtype>
void GMMLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const int datacount = top[0]->count();
  const int batchsize = bottom[0]->shape()[0];
  if (M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                         weight, bottom_data, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[0],
                            this->blobs_[1]->gpu_data(), top_data);
  }
  else {
	  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
		  bottom_data, weight, (Dtype)0., top_data);
	  if (bias_term_)
		  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
		  bias_multiplier_.gpu_data(),
		  this->blobs_[1]->gpu_data(), (Dtype)1., top_data);
  }

  //inner product ���� Gaussian mixture parameter calculate
  //0: alpha, 1~x : mu, x+1 : sigma 

  ///////////////////
  Dtype box[55];

  //find alpha max
  kernel_alpha_max<Dtype> << <CAFFE_GET_BLOCKS(batchsize), CAFFE_CUDA_NUM_THREADS >> >(batchsize, data_dim+2, class_size, top_data, maxValue_.mutable_gpu_data());

  //sub alpha max
  kernel_alpha_subtract<Dtype> << <CAFFE_GET_BLOCKS(class_size * batchsize), CAFFE_CUDA_NUM_THREADS >> >(class_size * batchsize, data_dim + 2, class_size, maxValue_.gpu_data(), top_data);

  //exponential - sigma�� exp�� ���ϴ� ���� ������ �� �� ����. overflow. ( alpha�� ������ sub max�� �������� overflow ����) ==> ������ ���� ��� 1/sigma �� ���� ==> 1/sigma�� ����
  sigmaExp<Dtype> << <CAFFE_GET_BLOCKS(datacount), CAFFE_CUDA_NUM_THREADS >> >(datacount, data_dim+2, sigma_min, sigma_max, top_data);

  //sum alpha
  kernel_alpha_sum<Dtype> << <CAFFE_GET_BLOCKS(batchsize), CAFFE_CUDA_NUM_THREADS >> >(batchsize, data_dim + 2, class_size, top_data, maxValue_.mutable_gpu_data());

  //div alpha
  kernel_alpha_div<Dtype> << <CAFFE_GET_BLOCKS(class_size * batchsize), CAFFE_CUDA_NUM_THREADS >> >(class_size * batchsize, data_dim + 2, class_size, maxValue_.gpu_data(), top_data);
}

template <typename Dtype>
void GMMLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (this->param_propagate_down_[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();

    // Gradient with respect to weight
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
        top_diff, bottom_data, (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bias
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
        bias_multiplier_.gpu_data(), (Dtype)1.,
        this->blobs_[1]->mutable_gpu_diff());
  }
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bottom data
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
        top_diff, this->blobs_[0]->gpu_data(), (Dtype)0.,
        bottom[0]->mutable_gpu_diff());
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(GMMLayer);

}  // namespace caffe
