#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/depth_concat_layer.hpp"

#include <opencv2\opencv.hpp>

namespace caffe {

	template <typename Dtype>
	__global__ void kernel_features_concat(const int num, const int total_dim, const int dim_1,
		const int dim_2, const Dtype* data_1, const Dtype* data_2,
		Dtype* out) {
		CUDA_KERNEL_LOOP(index, num) {
			int batch_idx = index / total_dim;
			int inner_idx = index % total_dim;

			if (inner_idx < dim_1){
				out[index] = data_1[batch_idx * dim_1 + inner_idx];
			}
			else{
				out[index] = data_2[batch_idx * dim_2 + (inner_idx - dim_1)];
			}
		}
	}

	template <typename Dtype>
	__global__ void kernel_features_seperate(const int num, const int total_dim, const int dim_1,
		const int dim_2, const Dtype* data, Dtype* out_1, Dtype* out_2) {
		CUDA_KERNEL_LOOP(index, num) {
			int batch_idx = index / total_dim;
			int inner_idx = index % total_dim;

			if (inner_idx < dim_1){
				out_1[batch_idx * dim_1 + inner_idx] = data[index];
			}
			else{
				out_2[batch_idx * dim_2 + (inner_idx - dim_1)] = data[index];
			}
		}
	}

	template <typename Dtype>
	void DepthConcatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		Dtype* top_data = top[0]->mutable_gpu_data();
		const Dtype* first_bot_data = bottom[0]->gpu_data();
		const Dtype* second_bot_data = bottom[1]->gpu_data();
		int count = top[0]->count();

		kernel_features_concat<Dtype> << <CAFFE_GET_BLOCKS(count),
			CAFFE_CUDA_NUM_THREADS >> >(count, first_dim + second_dim, first_dim, second_dim,
			first_bot_data, second_bot_data, top_data);
	}

	template <typename Dtype>
	void DepthConcatLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* first_bot_diff = bottom[0]->mutable_gpu_diff();
		Dtype* second_bot_diff = bottom[1]->mutable_gpu_diff();
		int count = top[0]->count();

		kernel_features_seperate<Dtype> << <CAFFE_GET_BLOCKS(count),
			CAFFE_CUDA_NUM_THREADS >> >(count, first_dim + second_dim, first_dim, second_dim,
			top_diff, first_bot_diff, second_bot_diff);
	}

	INSTANTIATE_LAYER_GPU_FUNCS(DepthConcatLayer);

}  // namespace caffe
