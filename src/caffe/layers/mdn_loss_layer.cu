#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/mdn_loss_layer.hpp"

#include <opencv2\opencv.hpp>

#define MATH_PI		3.14159265

namespace caffe {

template <typename Dtype>							//mu_ik - tk calculation
__global__ void kernel_label_subtract(const int count,
	const int param_size, const int class_size, const int data_dim,
	const Dtype* data, const Dtype* label, Dtype* diff) {
	CUDA_KERNEL_LOOP(index, count) {
		int internal_idx = index % data_dim;					//mu vector���� ���° �ε���
		int outer_idx = index / data_dim;						//���° Ŭ����
		int label_idx = index / (class_size * data_dim);		//���° label == ���° batch
		diff[index] = data[outer_idx * param_size + internal_idx + 1] - label[label_idx * data_dim + internal_idx];
	}
}

template <typename Dtype>							// || mu-t || ^ 2
__global__ void kernel_diff_norm(const int count,
	const int class_size, const int data_dim,
	const Dtype* diff_squre, Dtype* norm) {
	CUDA_KERNEL_LOOP(index, count) {
		Dtype sum = 0;
		for (int i = 0; i < data_dim; i++)
			sum += diff_squre[index * data_dim + i];
		norm[index] = sum;
	}
}

template <typename Dtype>							// alpha * gaussian distribution ���
__global__ void kernel_normal_distribution(const int count,
	const int param_size, const int class_size, const int data_dim,
	const Dtype* norm, const Dtype* data, Dtype* alpha_distribution) {
	CUDA_KERNEL_LOOP(index, count) {
		Dtype alpha = data[index*param_size];
		Dtype sigma = data[index*param_size + 1 + data_dim];
		Dtype sigma_9 = pow(sigma, data_dim);
		Dtype exp_gaussian = exp(- norm[index] / sigma / sigma / 2);
		Dtype distribution = 0;
		if (exp_gaussian != 0)
			distribution = exp_gaussian / pow(sigma, data_dim) / pow(2 * MATH_PI, data_dim / 2);
		//alpha * gaussian_distribution;
		alpha_distribution[index] = alpha * distribution;
	}
}

template <typename Dtype>							// ��(alpha * gaussian distribution) ���
__global__ void kernel_class_summation(const int count, const int class_size,
	const Dtype* alpha_pi_, Dtype* alpha_pi_sum_) {
	CUDA_KERNEL_LOOP(index, count) {
		Dtype sum = 0;
		for (int i = 0; i < class_size; i++)
			sum += alpha_pi_[index * class_size + i];
		if (sum != 0)
			alpha_pi_sum_[index] = sum;
	}
}

template <typename Dtype>							// posterior calculation 
__global__ void kernel_posterior_calc(const int count,
	const int batch_size, const int class_size,
	const Dtype* alpha_pi_, const Dtype* alpha_pi_sum_, Dtype* posterior) {
	CUDA_KERNEL_LOOP(index, count) {
		const int batch_idx = index / class_size;
		posterior[index] = alpha_pi_[index] / alpha_pi_sum_[batch_idx];
	}
}

template <typename Dtype>							// backpropagation delta calculation 
__global__ void kernel_delta_calc(const int count,
	const int batch_size, const int class_size, const int param_size, const int data_dim, float sigma_min, float sigma_max,
 	const Dtype* posterior, const Dtype* diff, const Dtype* diff_norm, const Dtype* bottom_data, Dtype* bottom_diff) {
	CUDA_KERNEL_LOOP(index, count) {
 		const int internal_idx = index % param_size;
		const int class_idx = index / param_size;
		const Dtype sigma = bottom_data[class_idx*param_size + param_size - 1];
		if (internal_idx == 0){							//alpha delta calculate
			bottom_diff[index] = bottom_data[index] - posterior[class_idx];
		}
		else if (internal_idx == param_size - 1){		//sigma delta calculate
			if (sigma_min > sigma || sigma_max < sigma)		bottom_diff[index] = 0;
			else
				bottom_diff[index] = -posterior[class_idx] * (diff_norm[class_idx] / sigma / sigma - data_dim);
		}
		else{											//mu delta calculate
			const int data_idx = internal_idx - 1;		//[0, datadim-1]
			Dtype diff_ik = diff[data_dim * class_idx + data_idx];
			bottom_diff[index] = posterior[class_idx] * (diff_ik / sigma / sigma);
		}
	}
}

//template <typename Dtype>
//__global__ void kernel_gradient_cliping(const int count,
//	const int batch_size, const int class_size, const int class_dim,
//	const Dtype threshold, const Dtype* norm, Dtype* gradient ){
//	CUDA_KERNEL_LOOP(index, count) {
//		const int batchIdx = index % (class_size * class_dim);
//
//		if (norm[batchIdx] > threshold){
//			gradient[index] = threshold / norm[index] * gradient[index];
//		}
//	}
//}
//
//template <typename Dtype>
//__global__ void kernel_gradient_norm(const int count,
//	const int batch_size, const int class_size, const int class_dim,
//	const Dtype *squre, Dtype *norm){
//	CUDA_KERNEL_LOOP(index, count) {
//		const int batchidx = index;
//
//		Dtype sum = 0;
//		for (int i = 0; i < class_dim * class_size; i++){
//			sum += squre[batchidx * (class_dim * class_size) + i];
//		}
//		norm[batchidx] = sqrt(sum);
//	}
//}

template <typename Dtype>
__global__ void kernel_inexp_x(const int count,
	const int param_size, const int class_size, const int data_dim,
	const Dtype* norm, const Dtype* data, Dtype* alpha_pi_x){
	CUDA_KERNEL_LOOP(index, count) {
		const Dtype epsilon = 0.00001;
		Dtype alpha = data[index*param_size];
		Dtype sigma = data[index*param_size + 1 + data_dim];
		alpha_pi_x[index] = log(alpha + epsilon) - data_dim / 2 * log(2 * MATH_PI*sigma + epsilon) - norm[index] / 2 / sigma;
	}
}

template <typename Dtype>
__global__ void kernel_max_inexp(const int count, 
	const int batch_size, const int class_size,
	const Dtype* alpha_pi_x, Dtype* max){
	CUDA_KERNEL_LOOP(index, count) {
		max[index] = -FLT_MAX;
		for (int i = 0; i < class_size; i++){
			if (max[index] < alpha_pi_x[index*class_size + i])
				max[index] = alpha_pi_x[index*class_size + i];
		}
	}
}

template <typename Dtype>
__global__ void kernel_sub_inexp_m(const int count,
	const int batch_size, const int class_size,
	const Dtype * max, const Dtype *x,
	Dtype *x_m, Dtype *alpha_pi_x){
	CUDA_KERNEL_LOOP(index, count) {
		int batch_idx = index / class_size;
		x_m[index] = x[index] - max[batch_idx];
		alpha_pi_x[index] = exp(x_m[index]);
	}
}

template <typename Dtype>
__global__ void kernel_calc_inexp_loss(const int count,
	const int batch_size, const int class_size,
	const Dtype *max, const Dtype *alpha_pi_x, Dtype *alpha_pi_x_sum, Dtype *batch_loss){
	CUDA_KERNEL_LOOP(index, count) {
		alpha_pi_x_sum[index] = 0;	//��e^(x-m)
		for (int i = 0; i < 5; i++){
			alpha_pi_x_sum[index] += alpha_pi_x[class_size * index + i];
		}
		batch_loss[index] = max[index] + log(alpha_pi_x_sum[index]);
	}
}

template <typename Dtype>
void MDNLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

	const Dtype* bottom_data = bottom[0]->gpu_data();
	const Dtype* label = bottom[1]->gpu_data();
	const int batch_size = bottom[0]->shape()[0];

	//subtract (mu - t)
	kernel_label_subtract<Dtype> << <CAFFE_GET_BLOCKS(diff_.count()), CAFFE_CUDA_NUM_THREADS >> >(diff_.count(),
		data_dim + 2, class_size, data_dim, bottom_data, label, diff_.mutable_gpu_data());

	//square ( mu - t )^2
	caffe_gpu_mul(diff_.count(), diff_.gpu_data(), diff_.gpu_data(), diff_square_.mutable_gpu_data());

	//norm  : || mu-t || ^ 2
	kernel_diff_norm<Dtype> << <CAFFE_GET_BLOCKS(class_size * batch_size), CAFFE_CUDA_NUM_THREADS >> >(class_size * batch_size,
		class_size, data_dim, diff_square_.gpu_data(), diff_norm_.mutable_gpu_data());

	//calculate gaussian distribution
	//kernel_normal_distribution<Dtype> << <CAFFE_GET_BLOCKS(class_size * batch_size), CAFFE_CUDA_NUM_THREADS >> >(class_size * batch_size,
	//	data_dim + 2, class_size, data_dim,
	//	diff_norm_.gpu_data(), bottom_data, alpha_pi_.mutable_gpu_data());

	//find max_alpha_pi
	//calc x
	kernel_inexp_x<Dtype> << <CAFFE_GET_BLOCKS(class_size * batch_size), CAFFE_CUDA_NUM_THREADS >> >(class_size * batch_size,
		data_dim + 2, class_size, data_dim,
		diff_norm_.gpu_data(), bottom_data, x_.mutable_gpu_data());
	//find max
	kernel_max_inexp<Dtype> << <CAFFE_GET_BLOCKS(batch_size), CAFFE_CUDA_NUM_THREADS >> >(batch_size,
		batch_size, class_size,
		x_.gpu_data(), max_alpha_pi_.mutable_gpu_data());
	//sub exp(x-m)
	kernel_sub_inexp_m<Dtype> << <CAFFE_GET_BLOCKS(class_size * batch_size), CAFFE_CUDA_NUM_THREADS >> >(class_size * batch_size,
		batch_size, class_size,
		max_alpha_pi_.gpu_data(), x_.gpu_data(),
		x_m_.mutable_gpu_data(), alpha_pi_.mutable_gpu_data());
	//calc loss
	kernel_calc_inexp_loss<Dtype> << <CAFFE_GET_BLOCKS(batch_size), CAFFE_CUDA_NUM_THREADS >> >(batch_size,
		batch_size, class_size,
		max_alpha_pi_.gpu_data(), alpha_pi_.gpu_data(), alpha_pi_sum_.mutable_gpu_data(), batch_loss_.mutable_gpu_data());

	//sumation : ��(alpha * distribution)
	//kernel_class_summation<Dtype> << <CAFFE_GET_BLOCKS(batch_size), CAFFE_CUDA_NUM_THREADS >> >(batch_size, class_size, alpha_pi_.gpu_data(), alpha_pi_sum_.mutable_gpu_data());

	//loss : ln ( sumation ) / number of batchsize
	Dtype loss;
	//caffe_gpu_log(alpha_pi_sum_.count(), alpha_pi_sum_.gpu_data(), batch_loss_.mutable_gpu_data());
	caffe_gpu_dot(batch_loss_.count(), batch_loss_.gpu_data(), sum_multiplier_.gpu_data(), &loss);
	loss /= bottom[0]->num();
	top[0]->mutable_cpu_data()[0] = -loss;

	if (visualize_){
		//write file
		FILE *fp = fopen("loss_output.txt", "w");

		Dtype *out = new Dtype[class_size * 11];
		Dtype label_out[9];
		hipMemcpy(out, bottom[0]->gpu_data(), sizeof(Dtype) * class_size * 11, hipMemcpyDeviceToHost);
		hipMemcpy(label_out, bottom[1]->gpu_data(), sizeof(Dtype) * 9, hipMemcpyDeviceToHost);

		for (int i = 0; i < 9; i++)
			fprintf(fp, "%f ", label_out[i]);
		fprintf(fp, "\n");

		for (int i = 0; i < class_size * 11; i++)
			fprintf(fp, "%f ", out[i]);

		delete[] out;

		fclose(fp);
	}

	if (std::isnan(loss) || std::isinf(loss) /*|| loop_count % 100 == 0*/){
		printf("loss invalid value.\n");
		loop_count = 0;

		Dtype norm_box[5];
		Dtype diff_box[45], diff_squre_box[45];
		Dtype bot_box[55], label_box[9];
		Dtype dist_box[5], m_box;
		Dtype x_box[5], x_m_box[5];
		Dtype norm;
		Dtype alpha_pi_sum__box, alpha_pi_sum__box_temp;
		Dtype lossslice;

		for (int i = 0; i < batch_size; i++){
			hipMemcpy(diff_box, &diff_.gpu_data()[i * 45], sizeof(Dtype) * 45, hipMemcpyDeviceToHost);
			hipMemcpy(label_box, &label[i * 9], sizeof(Dtype) * 9, hipMemcpyDeviceToHost);
			hipMemcpy(bot_box, &bottom_data[55 * i], sizeof(Dtype) * 55, hipMemcpyDeviceToHost);
			hipMemcpy(diff_squre_box, &diff_square_.gpu_data()[i * 45], sizeof(Dtype) * 45, hipMemcpyDeviceToHost);
			hipMemcpy(norm_box, &diff_norm_.gpu_data()[i * 5], sizeof(Dtype) * 5, hipMemcpyDeviceToHost);
			hipMemcpy(dist_box, &alpha_pi_.gpu_data()[i * 5], sizeof(Dtype) * 5, hipMemcpyDeviceToHost);
			hipMemcpy(&m_box, &max_alpha_pi_.gpu_data()[i], sizeof(Dtype), hipMemcpyDeviceToHost);
			hipMemcpy(x_box, &x_.gpu_data()[i * 5], sizeof(Dtype) * 5, hipMemcpyDeviceToHost);
			hipMemcpy(x_m_box, &x_m_.gpu_data()[i * 5], sizeof(Dtype) * 5, hipMemcpyDeviceToHost);
			hipMemcpy(&alpha_pi_sum__box, &alpha_pi_sum_.gpu_data()[i], sizeof(Dtype), hipMemcpyDeviceToHost);
			hipMemcpy(&lossslice, &batch_loss_.gpu_data()[i], sizeof(Dtype), hipMemcpyDeviceToHost);

			if (std::isnan(lossslice) || std::isinf(lossslice)){
				printf("slice of loss overflow\n");

				Dtype TotalX[5 * 128];
				hipMemcpy(TotalX, x_.gpu_data(), sizeof(Dtype) * 128 * 5, hipMemcpyDeviceToHost);
				for (int index = 0; index < 128; index++){
					Dtype tempmax = -FLT_MAX;
					for (int i = 0; i < class_size; i++){
						if (tempmax < TotalX[index*class_size + i])
							tempmax = TotalX[index*class_size + i];
					}
				}
			}
		}
	}

	//loop_count++;
}

//Diff 0������ ���ְ� 1������ ����
template <typename Dtype>
void MDNLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	for (int i = 0; i < 2; ++i) {
		if (propagate_down[i]) {
			// i == 0 : bottom network i == 1 : label
			//��ȣ +- �ٽ� �ѹ� �����غ���
			Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
			const Dtype* bottom_data = bottom[i]->gpu_data();
			const int batch_size = bottom[0]->shape()[0];
			
			//calculate posterior probability ( alpha*pi / sumation ( alpha_i * pi_i )
			kernel_posterior_calc<Dtype> << <CAFFE_GET_BLOCKS(batch_size*class_size), CAFFE_CUDA_NUM_THREADS >> >
				(batch_size*class_size, batch_size, class_size, 
				alpha_pi_.gpu_data(), alpha_pi_sum_.gpu_data(), posterior_pi_.mutable_gpu_data());

			//calculate bottom diff (alpha_diff, mu_diff, sigma_diff)
			kernel_delta_calc<Dtype> << <CAFFE_GET_BLOCKS(bottom[i]->count()), CAFFE_CUDA_NUM_THREADS >> >(bottom[i]->count(),
				batch_size, class_size, data_dim + 2, data_dim, sigma_min, sigma_max,
				posterior_pi_.gpu_data(), diff_.gpu_data(), diff_norm_.gpu_data(), bottom_data, bottom_diff);

			//Dtype gradTemp[55];
			//for (int j = 0; j < batch_size; j++){
			//	hipMemcpy(gradTemp, &bottom[i]->mutable_gpu_diff()[j * 55], sizeof(Dtype) * 55, hipMemcpyDeviceToHost);

			//}
			
		}
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(MDNLossLayer);

}  // namespace caffe
