#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/mdn_loss_layer.hpp"

#include <opencv2\opencv.hpp>

#define MATH_PI		3.14159265

namespace caffe {

template <typename Dtype>							//mu_ik - tk calculation
__global__ void kernel_label_subtract(const int count,
	const int param_size, const int class_size, const int data_dim,
	const Dtype* data, const Dtype* label, Dtype* diff) {
	CUDA_KERNEL_LOOP(index, count) {
		int internal_idx = index % data_dim;					//mu vector���� ���° �ε���
		int outer_idx = index / data_dim;						//���° Ŭ����
		int label_idx = index / (class_size * data_dim);		//���° label == ���° batch
		diff[index] = data[outer_idx * param_size + internal_idx + 1] - label[label_idx * data_dim + internal_idx];
	}
}

template <typename Dtype>							// || mu-t || ^ 2
__global__ void kernel_diff_norm(const int count,
	const int class_size, const int data_dim,
	const Dtype* diff_squre, Dtype* norm) {
	CUDA_KERNEL_LOOP(index, count) {
		Dtype sum = 0;
		for (int i = 0; i < data_dim; i++)
			sum += diff_squre[index * data_dim + i];
		norm[index] = sum;
	}
}

template <typename Dtype>							// alpha * gaussian distribution ���
__global__ void kernel_normal_distribution(const int count,
	const int param_size, const int class_size, const int data_dim,
	const Dtype* norm, const Dtype* data, Dtype* alpha_distribution) {
	CUDA_KERNEL_LOOP(index, count) {
		Dtype alpha = data[index*param_size];
		Dtype sigma = data[index*param_size + 1 + data_dim];
		Dtype sigma_9 = pow(sigma, data_dim);
		Dtype exp_gaussian = exp(- norm[index] / sigma / sigma / 2);
		Dtype distribution = 0;
		if (exp_gaussian != 0)
			distribution = exp_gaussian / pow(sigma, data_dim) / pow(2 * MATH_PI, data_dim / 2);
		//alpha * gaussian_distribution;
		alpha_distribution[index] = alpha * distribution;
	}
}

template <typename Dtype>							// ��(alpha * gaussian distribution) ���
__global__ void kernel_class_summation(const int count, const int class_size,
	const Dtype* alpha_pi_, Dtype* alpha_pi_sum_) {
	CUDA_KERNEL_LOOP(index, count) {
		Dtype sum = 0;
		for (int i = 0; i < class_size; i++)
			sum += alpha_pi_[index * class_size + i];
		if (sum != 0)
			alpha_pi_sum_[index] = sum;
	}
}

template <typename Dtype>							// posterior calculation 
__global__ void kernel_posterior_calc(const int count,
	const int batch_size, const int class_size,
	const Dtype* alpha_pi_, const Dtype* alpha_pi_sum_, Dtype* posterior) {
	CUDA_KERNEL_LOOP(index, count) {
		const int batch_idx = index / class_size;
		posterior[index] = alpha_pi_[index] / alpha_pi_sum_[batch_idx];
	}
}

template <typename Dtype>							// backpropagation delta calculation 
__global__ void kernel_delta_calc(const int count,
	const int batch_size, const int class_size, const int param_size, const int data_dim, float sigma_min, float sigma_max,
 	const Dtype* posterior, const Dtype* diff, const Dtype* diff_norm, const Dtype* bottom_data, Dtype* bottom_diff) {
	CUDA_KERNEL_LOOP(index, count) {
 		const int internal_idx = index % param_size;
		const int class_idx = index / param_size;
		const Dtype sigma = bottom_data[class_idx*param_size + param_size - 1];
		if (internal_idx == 0){							//alpha delta calculate
			bottom_diff[index] = bottom_data[index] - posterior[class_idx];
		}
		else if (internal_idx == param_size - 1){		//sigma delta calculate
			if (sigma_min > sigma || sigma_max < sigma)		bottom_diff[index] = 0;
			else
				bottom_diff[index] = -posterior[class_idx] * (diff_norm[class_idx] / sigma / sigma - data_dim);
		}
		else{											//mu delta calculate
			const int data_idx = internal_idx - 1;		//[0, datadim-1]
			Dtype diff_ik = diff[data_dim * class_idx + data_idx];
			bottom_diff[index] = posterior[class_idx] * (diff_ik / sigma / sigma);
		}
	}
}

template <typename Dtype>
void MDNLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

	const Dtype* bottom_data = bottom[0]->gpu_data();
	const Dtype* label = bottom[1]->gpu_data();
	const int batch_size = bottom[0]->shape()[0];

	//subtract (mu - t)
	kernel_label_subtract<Dtype> << <CAFFE_GET_BLOCKS(diff_.count()), CAFFE_CUDA_NUM_THREADS >> >(diff_.count(),
		data_dim + 2, class_size, data_dim, bottom_data, label, diff_.mutable_gpu_data());

	//square ( mu - t )^2
	caffe_gpu_mul(diff_.count(), diff_.gpu_data(), diff_.gpu_data(), diff_square_.mutable_gpu_data());

	//norm  : || mu-t || ^ 2
	kernel_diff_norm<Dtype> << <CAFFE_GET_BLOCKS(class_size * batch_size), CAFFE_CUDA_NUM_THREADS >> >(class_size * batch_size,
		class_size, data_dim, diff_square_.gpu_data(), diff_norm_.mutable_gpu_data());

	//calculate gaussian distribution
	kernel_normal_distribution<Dtype> << <CAFFE_GET_BLOCKS(class_size * batch_size), CAFFE_CUDA_NUM_THREADS >> >(class_size * batch_size,
		data_dim + 2, class_size, data_dim,
		diff_norm_.gpu_data(), bottom_data, alpha_pi_.mutable_gpu_data());

	//sumation : ��(alpha * distribution)
	kernel_class_summation<Dtype> << <CAFFE_GET_BLOCKS(batch_size), CAFFE_CUDA_NUM_THREADS >> >(batch_size, class_size, alpha_pi_.gpu_data(), alpha_pi_sum_.mutable_gpu_data());

	//loss : ln ( sumation ) / number of batchsize
	Dtype loss;
	caffe_gpu_log(alpha_pi_sum_.count(), alpha_pi_sum_.gpu_data(), batch_loss_.mutable_gpu_data());
	caffe_gpu_dot(batch_loss_.count(), batch_loss_.gpu_data(), sum_multiplier_.gpu_data(), &loss);
	loss /= bottom[0]->num();
	top[0]->mutable_cpu_data()[0] = -loss;

	if (visualize_){
		//write file
		FILE *fp = fopen("loss_output.txt", "w");

		Dtype *out = new Dtype[class_size * 11];
		Dtype label_out[9];
		hipMemcpy(out, bottom[0]->gpu_data(), sizeof(Dtype) * class_size * 11, hipMemcpyDeviceToHost);
		hipMemcpy(label_out, bottom[1]->gpu_data(), sizeof(Dtype) * 9, hipMemcpyDeviceToHost);

		for (int i = 0; i < 9; i++)
			fprintf(fp, "%f ", label_out[i]);
		fprintf(fp, "\n");

		for (int i = 0; i < class_size * 11; i++)
			fprintf(fp, "%f ", out[i]);

		delete[] out;

		fclose(fp);
	}

	if (std::isnan(loss) || std::isinf(loss)){
		printf("loss invalid value.\n");

		Dtype norm_box[5];
		Dtype diff_box[45], diff_squre_box[45];
		Dtype bot_box[55], label_box[9];
		Dtype dist_box[5];
		Dtype sub;
		Dtype norm;
		Dtype alpha_pi_sum__box, alpha_pi_sum__box_temp;
		Dtype lossslice;
		for (int i = 0; i < batch_size; i++){
			hipMemcpy(diff_box, &diff_.gpu_data()[i * 45], sizeof(Dtype) * 45, hipMemcpyDeviceToHost);
			hipMemcpy(label_box, &label[i * 9], sizeof(Dtype) * 9, hipMemcpyDeviceToHost);
			hipMemcpy(bot_box, &bottom_data[55 * i], sizeof(Dtype) * 55, hipMemcpyDeviceToHost);
			hipMemcpy(diff_squre_box, &diff_square_.gpu_data()[i * 45], sizeof(Dtype) * 45, hipMemcpyDeviceToHost);
			hipMemcpy(norm_box, &diff_norm_.gpu_data()[i * 5], sizeof(Dtype) * 5, hipMemcpyDeviceToHost);
			hipMemcpy(dist_box, &alpha_pi_.gpu_data()[i * 5], sizeof(Dtype) * 5, hipMemcpyDeviceToHost);
			hipMemcpy(&alpha_pi_sum__box, &alpha_pi_sum_.gpu_data()[i], sizeof(Dtype), hipMemcpyDeviceToHost);
			hipMemcpy(&lossslice, &batch_loss_.gpu_data()[i], sizeof(Dtype), hipMemcpyDeviceToHost);

			if (std::isnan(lossslice) || std::isinf(lossslice))
				printf("slice of loss overflow\n");

			for (int j = 0; j < 70; j++)
				if (std::isnan(bot_box[j]) || std::isinf(bot_box[j]))
					printf("bottom data overflow.\n");
			//for (int j = 0; j < 45; j++){
			//	int tClass_idx = j / 9;
			//	int internal_idx = j % 9;
			//	sub = bot_box[tClass_idx * 11 + internal_idx + 1] - label_box[j % 9];

			//	if (diff_box[j] != sub){
			//		printf("diff miss\n");
			//	}

			//	if (diff_squre_box[j] != (sub*sub))
			//		printf("square miss\n");

			//	if (std::isnan(diff_box[j]) || std::isinf(diff_box[j]))
			//		printf("diff_box data overflow.\n");

			//}

			for (int j = 0; j < 5; j++){
				norm = 0;
				for (int k = 0; k < 12; k++)
					norm += diff_squre_box[j * 12 + k];
				if (norm != norm_box[j])
					printf("norm error\n");
																																	
			}
			for (int j = 0; j < 5; j++){
				Dtype alpha = bot_box[14 * j];
				Dtype sigma = bot_box[14 * j + 13];
				float exp_gaussian = exp(-norm_box[j] / sigma / sigma / 2);
				float sigma_9 = pow(sigma, 9);
				float pi_squre = pow(2 * MATH_PI, -9 / 2);
				float gaussian = 0;
				if(exp_gaussian !=0)
					gaussian = exp_gaussian / sigma_9 / pi_squre;
				Dtype dist_temp = alpha * gaussian;
				if (std::isnan(dist_box[j]) || std::isinf(dist_box[j]) || dist_box[j] < 0)
					printf("norm_box data overflow.\n");

			}

			alpha_pi_sum__box_temp = 0;
			for (int j = 0; j < 10; j++){
				alpha_pi_sum__box_temp += dist_box[j];

			}
			if (alpha_pi_sum__box_temp != alpha_pi_sum__box)
				printf("summation error");
		}
	}
}

//Diff 0������ ���ְ� 1������ ����
template <typename Dtype>
void MDNLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	for (int i = 0; i < 2; ++i) {
		if (propagate_down[i]) {
			// i == 0 : bottom network i == 1 : label
			//��ȣ +- �ٽ� �ѹ� �����غ���
			Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
			const Dtype* bottom_data = bottom[i]->gpu_data();
			const int batch_size = bottom[0]->shape()[0];
			
			//calculate posterior probability ( alpha*pi / sumation ( alpha_i * pi_i )
			kernel_posterior_calc<Dtype> << <CAFFE_GET_BLOCKS(batch_size*class_size), CAFFE_CUDA_NUM_THREADS >> >
				(batch_size*class_size, batch_size, class_size, 
				alpha_pi_.gpu_data(), alpha_pi_sum_.gpu_data(), posterior_pi_.mutable_gpu_data());

			//calculate bottom diff (alpha_diff, mu_diff, sigma_diff)
			kernel_delta_calc<Dtype> << <CAFFE_GET_BLOCKS(bottom[i]->count()), CAFFE_CUDA_NUM_THREADS >> >(bottom[i]->count(),
				batch_size, class_size, data_dim + 2, data_dim, sigma_min, sigma_max,
				posterior_pi_.gpu_data(), diff_.gpu_data(), diff_norm_.gpu_data(), bottom_data, bottom_diff);
		}
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(MDNLossLayer);

}  // namespace caffe
