#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/spatial_softmax_layer.hpp"

#include <opencv2\opencv.hpp>

namespace caffe {

template <typename Dtype>
__global__ void kernel_feature_max(const int num, const int width,
	const int height, const Dtype* data, Dtype* out) {
	CUDA_KERNEL_LOOP(index, num) {
		Dtype maxval = -FLT_MAX;
		for (int i = 0; i < height * width; i++)
			maxval = max(data[(index*width*height) + i], maxval);

		out[index] = maxval + 0.0000001;
	}
}

template <typename Dtype>
__global__ void kernel_feature_subtract(const int count,
	const int width, const int height,
	const Dtype* feature_max, Dtype* data) {
	CUDA_KERNEL_LOOP(index, count) {
		int n = index / width / height;
		data[index] -= feature_max[n];
	}
}

template <typename Dtype>
__global__ void kernel_exp(const int count, const Dtype* data, Dtype* out) {
	CUDA_KERNEL_LOOP(index, count) {
		out[index] = exp(data[index]);
	}
}

template <typename Dtype>
__global__ void kernel_features_scale(const int count,
	const int width, const int height,
	const Dtype* max, Dtype scaleparam, Dtype* data) {
	CUDA_KERNEL_LOOP(index, count) {
		int n = index / width / height;
		data[index] = data[index] * scaleparam /*/ max[n]*/;
		//if (max[n] != 0)
		//	data[index] = data[index] * scaleparam /*/ max[n]*/;
	}
}

template <typename Dtype>
__global__ void kernel_features_div(const int count,
	const int width, const int height,
	const Dtype* sum, Dtype* data) {
	CUDA_KERNEL_LOOP(index, count) {
		int n = index / width / height;
		data[index] /= sum[n];
	}
}

template <typename Dtype>
__global__ void kernel_feature_sum(const int num, const int width,
	const int height, const Dtype* data, Dtype* out) {
	CUDA_KERNEL_LOOP(index, num) {
		const Dtype* const bottom_slice =
			data + index * height * width;						//softmax���� �� ���ĸ��� �߶�.(ä���� 1�̱� ������ ä���� ���X)

		Dtype sum = 0;
		for (int h = 0; h < height; h++){
			for (int w = 0; w < width; w++)
				sum += bottom_slice[h*width + w];
		}

		out[index] = sum;
	}
}

template <typename Dtype>
__global__ void SpatialForward(const int nthreads,
	const Dtype* const bottom_data, const int num,
	const int height, const int width, Dtype* const top_data) {
	CUDA_KERNEL_LOOP(index, nthreads) {							//CUDA KERNEL LOOP �Լ��� index�� nthread���� ������ for ���� - �������� �����ؾ���.
		
		//���� ����
		Dtype tValue = 0.0f;

		for (int h = 0; h < height; h++){									
			for (int w = 0; w < width; w++){
				const int element_idx = h * width + w;						//������ �ؾ��� element
				if (index % 2 == 0)										//¦���϶��� x��ǥ ���õ� �۾�
					tValue += w * bottom_data[(index / 2)*width*height + element_idx] / (Dtype)width;
				else														//Ȧ���϶��� y��ǥ ���õ� �۾�
					tValue += h * bottom_data[(index / 2)*width*height + element_idx] / (Dtype)height;
			}
		}

		top_data[index] = tValue;
	}
}

template <typename Dtype>
__global__ void SpatialBackward(const int nthreads,
	const Dtype* const top_diff, const int batchSize, const int nChannels,
	const int bottom_height, const int bottom_width, Dtype* const bottom_diff) {
	CUDA_KERNEL_LOOP(index, nthreads) {							//CUDA KERNEL LOOP �Լ��� index�� nthread���� ������ for ���� - �������� �����ؾ���.
		//index = 
		const int mapidx = index / (bottom_height * bottom_width);				//���° ������ ���
		const int inMapidx = index % (bottom_height * bottom_width);			//map �ȿ��� ���° �ε��� ����
		const int w = inMapidx % bottom_width;
		const int h = inMapidx / bottom_width;
		bottom_diff[index] = (w * top_diff[2 * mapidx + 0] / (Dtype)bottom_width)
							+ (h * top_diff[2 * mapidx + 1] / (Dtype)bottom_height);
	}
}

template <typename Dtype>
__global__ void kernel_features_dot(const int num, const int width,
	const int height, const Dtype* data_1, const Dtype* data_2,
	Dtype* out) {
	CUDA_KERNEL_LOOP(index, num) {
		Dtype result = 0;
		int mapIdx = index*width*height;

		for (int i = 0; i < width*height; i++)
			result += data_1[mapIdx + i] * data_2[mapIdx + i];

		out[index] = result;
	}
}

template <typename Dtype>
void SpatialSoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
	//Dtype* bottom_data = bottom[0]->mutable_gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int tWidth = bottom[0]->shape()[2];
  int tHeight = bottom[0]->shape()[3];

  /*int sTime = clock();*/

  //////////////////////////////////softmax operation///////////////////////////////////////////////////// check
  //find max
  const int feature_count = bottom[0]->count() / (tWidth*tHeight);						//�� ���� 32*60
  Dtype* max_data = maxValue_.mutable_gpu_data();
  Dtype* softmax_result = softmaxResult_.mutable_gpu_data();
  const int softmaxCount = softmaxResult_.count();
  caffe_copy(bottom[0]->count(), bottom_data, softmax_result);

  if (alpha_ != 1.0f){
	  kernel_features_scale<Dtype> << <CAFFE_GET_BLOCKS(softmaxCount),
		   CAFFE_CUDA_NUM_THREADS >> >(softmaxCount, tWidth, tHeight,
		   max_data, (Dtype)alpha_, softmax_result);
  }

  kernel_feature_max<Dtype> << <CAFFE_GET_BLOCKS(feature_count),
	  CAFFE_CUDA_NUM_THREADS >> >(feature_count, tWidth, tHeight, softmax_result,
	  max_data);

  ////sub max
  kernel_feature_subtract<Dtype> << <CAFFE_GET_BLOCKS(softmaxCount),
	  CAFFE_CUDA_NUM_THREADS >> >(softmaxCount, tWidth, tHeight,
	  max_data, softmax_result);

  //exponential
  kernel_exp<Dtype> << <CAFFE_GET_BLOCKS(softmaxCount), CAFFE_CUDA_NUM_THREADS >> >(
	  softmaxCount, softmax_result, softmax_result);

  //calculate summation
  kernel_feature_sum<Dtype> << <CAFFE_GET_BLOCKS(feature_count),
	  CAFFE_CUDA_NUM_THREADS >> >(feature_count, tWidth, tHeight, softmax_result,
	  backwardTemp_.mutable_gpu_data());

  //div result
  kernel_features_div<Dtype> << <CAFFE_GET_BLOCKS(softmaxCount),
	  CAFFE_CUDA_NUM_THREADS >> >(softmaxCount, tWidth, tHeight, backwardTemp_.gpu_data(), softmax_result);

  //////////////////////////////////extract feature postion///////////////////////////////////////////////
  //<<���� ��ϼ�, �ڴ� ��ϴ� �������>>
  const Dtype* softmax = softmaxResult_.gpu_data();
  int count = top[0]->count();
  SpatialForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
	  count, softmax, softmaxResult_.num(),
	  tHeight, tWidth, top_data);

  //Feature �Ѹ���
  //////if (bottom->size() == 2){
  if (is_visualize){
	  cv::Mat FeaturePlot;
	  const int drawRow = 10;
	  const int datatWidth = bottom[1]->shape()[2];
	  const int dataHeight = bottom[1]->shape()[3];
	  const int dataCount = bottom[1]->shape()[0];
	  const int softChannel = bottom[0]->shape()[1];
	  const int topCount = top[0]->shape()[1];
	  FeaturePlot.create(datatWidth * (dataCount / drawRow + 1), dataHeight * 2 * drawRow, CV_8UC3);
	  for (int i = 0; i < dataCount; i++){
		  int s_row = i * 2 / (drawRow * 2) * datatWidth;
		  int s_col = i * 2 % (drawRow * 2) * datatWidth;

		  Dtype pos[128];
		  Dtype *Map = new Dtype[160 * 160 * 3];
		  cv::Mat SigleFeature(160, 160, CV_8UC3);
		  cv::Point pointList[64];

		  hipMemcpy(pos, &top[0]->gpu_data()[i * topCount], sizeof(Dtype) * topCount, hipMemcpyDeviceToHost);
		  hipMemcpy(Map, &bottom[1]->gpu_data()[i * datatWidth * dataHeight * 3], sizeof(Dtype) * datatWidth * dataHeight * 3, hipMemcpyDeviceToHost);

		  for (int i = 0; i < topCount / 2; i++)
			  pointList[i] = cv::Point(pos[2 * i] * datatWidth, pos[2 * i + 1] * dataHeight);

		  for (int h = 0; h < dataHeight; h++){
			  for (int w = 0; w < datatWidth; w++){
				  for (int c = 0; c < 3; c++){
					  SigleFeature.at<cv::Vec3b>(h, w)[c] = uchar(Map[c*dataHeight*datatWidth + datatWidth*h + w] * 255.f);
					  FeaturePlot.at<cv::Vec3b>(s_row + h, s_col + w)[c] = uchar(Map[c*dataHeight*datatWidth + datatWidth*h + w] * 255.f);
					  FeaturePlot.at<cv::Vec3b>(s_row + h, s_col + w + datatWidth)[c] = uchar(Map[c*dataHeight*datatWidth + datatWidth*h + w] * 255.f);
				  }
			  }
		  }

		  Dtype softThreshold = 0.0;
		  const int softwidth = softmaxResult_.shape()[2];
		  const int softheight = softmaxResult_.shape()[3];
		  Dtype *softmap = new Dtype[softwidth * softheight];
		  for (int j = 0; j < topCount / 2 / 2; j++){
			  uchar R = (j * 7) % 255;
			  uchar G = (j * 37) % 255;
			  uchar B = (j * 103) % 255;

			  hipMemcpy(softmap, &softmaxResult_.gpu_data()[i*softChannel*softwidth*softheight + softwidth * softheight * j], sizeof(Dtype) * softwidth * softheight, hipMemcpyDeviceToHost);
			  Dtype Max = -99999;
			  for (int s = 0; s < softwidth*softheight; s++)
				  if (Max < softmap[s])
					  Max = softmap[s];

			  if (Max > softThreshold){
				  cv::circle(FeaturePlot, cv::Point(s_col + pointList[j].x, s_row + pointList[j].y), 3, cv::Scalar(B, G, R), -1);
				  cv::circle(SigleFeature, cv::Point(pointList[j].x, pointList[j].y), 3, cv::Scalar(B, G, R), -1);
			  }
		  }
		  for (int j = topCount / 2 / 2; j < topCount; j++){
			  uchar R = (j * 7) % 255;
			  uchar G = (j * 37) % 255;
			  uchar B = (j * 103) % 255;

			  hipMemcpy(softmap, &softmaxResult_.gpu_data()[i * softChannel * softwidth*softheight + softwidth * softheight * j], sizeof(Dtype) * softwidth * softheight, hipMemcpyDeviceToHost);
			  Dtype Max = -99999;
			  for (int s = 0; s < softwidth*softheight; s++)
				  if (Max < softmap[s])
					  Max = softmap[s];

			  if (Max > softThreshold)
				  cv::circle(FeaturePlot, cv::Point(s_col + datatWidth + pointList[j].x, s_row + pointList[j].y), 3, cv::Scalar(B, G, R), -1);
		  }

		  char buf[256];
		  sprintf(buf, "%d.bmp", i);
		  cv::imwrite(buf, SigleFeature);

		  delete[] softmap;
		  delete[] Map;
	  }

	  cv::imwrite("FeaturePlot.bmp", FeaturePlot);
	  cv::imshow("Map", FeaturePlot);
	  cv::waitKey(0);
  }
  ////////}
}

template <typename Dtype>
void SpatialSoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
	if (!propagate_down[0]) {			//pooling layer���� ����
		return;
	}

	const Dtype* top_diff = top[0]->gpu_diff();
	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
	const int bottom_height = bottom[0]->shape()[2];
	const int bottom_width = bottom[0]->shape()[3];
	const int count = bottom[0]->count();
	const int batchSize = bottom[0]->shape()[0];
	const int nChannels = bottom[0]->shape()[1];
	SpatialBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
		count, top_diff, batchSize, nChannels, bottom_height, bottom_width, bottom_diff);

	//softmax backward
	bottom_diff = bottom[0]->mutable_gpu_diff();

	//top_data(softmaxResult_) * top_diff = max_data
	const int feature_count = count / (bottom_height * bottom_width);
	Dtype* backwardTemp = backwardTemp_.mutable_gpu_data();
	const Dtype* softmax_result = softmaxResult_.gpu_data();
	kernel_features_dot<Dtype> << <CAFFE_GET_BLOCKS(feature_count),
		CAFFE_CUDA_NUM_THREADS >> >(feature_count, bottom_height, bottom_width,
		bottom_diff, softmax_result, backwardTemp);

	//bottom_diff - max_data = bottom_diff
	kernel_feature_subtract<Dtype> << <CAFFE_GET_BLOCKS(count),
		CAFFE_CUDA_NUM_THREADS >> >(count, bottom_width, bottom_height,
		backwardTemp, bottom_diff);

	//bottom_diff * top_data = bottom_diff (elementwise product)
	caffe_gpu_mul<Dtype>(softmaxResult_.count(), bottom_diff, softmax_result, bottom_diff);

	if (alpha_ != 1.0f){
		kernel_features_scale<Dtype> << <CAFFE_GET_BLOCKS(softmaxResult_.count()),
			CAFFE_CUDA_NUM_THREADS >> >(softmaxResult_.count(), bottom_width, bottom_height,
			maxValue_.gpu_data(), (Dtype)alpha_, bottom_diff);
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(SpatialSoftmaxLayer);

}  // namespace caffe
