#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/depth_value_concat_layer.hpp"

#include <opencv2\opencv.hpp>

namespace caffe {

	template <typename Dtype>
	__global__ void spatial_depth_concat(const int count, const int width, const int height,
		const Dtype* spatial_pos, const Dtype* depthval, Dtype* topdata) {
		CUDA_KERNEL_LOOP(index, count) {
			const int Internal_idx = index % 3;
			const int Feature_idx = index / 3;

			//x pos
			if (Internal_idx == 0)
				topdata[index] = spatial_pos[2 * Feature_idx + 0];
			//y pos
			else if (Internal_idx == 1)
				topdata[index] = spatial_pos[2 * Feature_idx + 1];
			//depth val
			else if (Internal_idx == 2){
				const int d_x = width * spatial_pos[2 * Feature_idx + 0];
				const int d_y = height * spatial_pos[2 * Feature_idx + 1];
				topdata[index] = depthval[d_x + d_y * width] / 1000.f;
			}
		}
	}

	template <typename Dtype>
	__global__ void concat_spatial_backward(const int count,
		const Dtype* topdiff, Dtype* spatial) {
		CUDA_KERNEL_LOOP(index, count) {
			const int featureIdx = index / 2;
			const int id = index % 2;				//0 : xpos, 1 : ypos 2 : depthvalue

			const int topid = featureIdx * 3 + id;

			spatial[index] = topdiff[topid];
		}
	}

	template <typename Dtype>
	void DepthValueConcatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const int topcount = top[0]->count();
		const int tWidth = bottom[1]->shape()[1];
		const int tHeight = bottom[1]->shape()[2];

		const Dtype* spatialPos = bottom[0]->gpu_data();						//spatial feature
		const Dtype* depthImg = bottom[1]->gpu_data();							//Depth image

		//concatenation
		spatial_depth_concat<Dtype> << <CAFFE_GET_BLOCKS(topcount),
			CAFFE_CUDA_NUM_THREADS >> >(topcount, tWidth, tHeight,
			spatialPos, depthImg, top[0]->mutable_gpu_data());
	}

	template <typename Dtype>
	void DepthValueConcatLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		const Dtype* topdiff = top[0]->gpu_diff();
		Dtype* spatialDiff = bottom[0]->mutable_gpu_diff();

		const int spatialcount = bottom[0]->count();

		//sptial positon Layer�θ� diff�� �����������
		concat_spatial_backward<Dtype> << <CAFFE_GET_BLOCKS(spatialcount),
			CAFFE_CUDA_NUM_THREADS >> >(spatialcount, topdiff, spatialDiff);
	}

	INSTANTIATE_LAYER_GPU_FUNCS(DepthValueConcatLayer);

}  // namespace caffe
